#include "hip/hip_runtime.h"

#include <iostream>
#include <cstring>
#include <string>
using namespace std;

#define NUM_DATA 512

__global__ void vecAdd(int *a,int *b,int *c)
{
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main()
{
    int *a,*b,*c;
    int *d_a,*d_b,*d_c;
    
    int memSize = sizeof(int)*NUM_DATA;
    cout << "elements : " << NUM_DATA <<"\n";
   
    a = new int[NUM_DATA]; memset(a,0,memSize);
    b = new int[NUM_DATA]; memset(a,0,memSize);
    c = new int[NUM_DATA]; memset(a,0,memSize);

    for(int i = 0 ; i < NUM_DATA; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }
    
    hipMalloc(&d_a,memSize);
    hipMalloc(&d_b,memSize);
    hipMalloc(&d_c,memSize);

    hipMemcpy(d_a,a,memSize,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,memSize,hipMemcpyHostToDevice);
    
    vecAdd<<<1,NUM_DATA>>>(d_a,d_b,d_c);
    hipDeviceSynchronize();
    hipMemcpy(c,d_c,memSize,hipMemcpyDeviceToHost);
    
    bool result = true;
    for(int i = 0 ; i < NUM_DATA; i++)
    {
        if(a[i] + b[i] != c[i]){
            cout << "Gpu has error in vecAdd\n";
            result = false;
        }
    }

    if(result)
        cout << "GPU WORKS WELL \n";
    
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 
    delete[] a; delete[] b; delete[] c;
    return 0;
}
 
